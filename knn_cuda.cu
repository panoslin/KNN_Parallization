#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <limits>
#include <chrono>
#include <algorithm>
#include <queue>
#include "libarff/arff_parser.h"
#include "libarff/arff_data.h"
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/pair.h>


using namespace std;


__global__ void find_knn(
    float* d_train_matrix,
    float *d_test_matrix, 
    int* d_predicitons, 
    thrust::pair<int, float> *d_knn,
    int k
) {

}


// Calculates the distance between two instances
float distance(const float* instance_A, const float* instance_B, int num_attributes) {
    float sum = 0.0f;
    for (int i = 0; i < num_attributes - 1; ++i) { // Exclude the class label
        float diff = instance_A[i] - instance_B[i];
        sum += diff * diff;
    }
    return sqrt(sum);
}

// Candidate neighbor structure
struct Candidate {
    float distance;
    int class_label;
};

// Comparator for the priority queue (max-heap)
struct CandidateComparator {
    bool operator()(const Candidate& lhs, const Candidate& rhs) const {
        return lhs.distance < rhs.distance;
    }
};

vector<int> KNN(ArffData* train, ArffData* test, int k) {
    int num_classes = train->num_classes();
    int num_attributes = train->num_attributes();
    int train_num_instances = train->num_instances();
    int test_num_instances = test->num_instances();

    vector<int> predictions(test_num_instances);

    float* train_matrix = train->get_dataset_matrix();
    float* test_matrix = test->get_dataset_matrix();

    // 0. Defined dims
    // Define a 1D of grid of 1D of block
    int threadPerBlock = 1024;
    // assuming the max shared memory space is 24KB (actually it's 48KB, just to be safe)
    // each shared mem will store pairs of (int, float) representing the (class, distance)
    int trainInstancePerBlock = 24 * 1024 / 8;
    int blockPerTestInstance = (train_num_instances + trainInstancePerBlock - 1) / trainInstancePerBlock;
    int blockPerGrid = blockPerTestInstance * test_num_instances;
    int trainInstancePerThread = trainInstancePerBlock / threadPerBlock;

    // 1. init mem
    float* d_train_matrix, *d_test_matrix;

    hipMalloc(&d_train_matrix, sizeof(float) * num_attributes * train_num_instances);
    hipMalloc(&d_test_matrix, sizeof(float) * num_attributes * test_num_instances);

    int* d_predictions;
    hipMalloc(&d_predictions, sizeof(int) * test_num_instances);

    // store the knns for each local knn for each block for each test instance
    // each block will calculate the knn locally and write to this d_knn global memory
    thrust::pair<int, float> *d_knn;
    hipMalloc((void**)&d_knn, test_num_instances * blockPerTestInstance * k * sizeof(thrust::pair<int, float>));

    // 2. Copy to device
    hipMemcpy(d_train_matrix, train_matrix, sizeof(float) * num_attributes * train_num_instances, hipMemcpyHostToDevice);
    hipMemcpy(d_test_matrix, test_matrix, sizeof(float) * num_attributes * test_num_instances, hipMemcpyHostToDevice);
    
    // Set init value to d_predicitons
    hipMemset(d_predictions, 0, sizeof(int) * test_num_instances);

    // 4. Call kernel
    find_knn<<< blockPerGrid, threadPerBlock>>> (d_train_matrix, d_test_matrix, d_predictions, d_knn, k);

    // 5. Copy to host
    hipMemcpy(predictions.data(), d_predictions, sizeof(int) * test_num_instances, hipMemcpyDeviceToHost);
    
    // 6. Free memory
    hipFree(d_train_matrix);
    hipFree(d_test_matrix);
    hipFree(d_predictions);
    hipFree(d_knn);

    return predictions;
}

vector<int> computeConfusionMatrix(const vector<int>& predictions, ArffData* dataset) {
    int num_classes = dataset->num_classes();
    int num_instances = dataset->num_instances();
    vector<int> confusionMatrix(num_classes * num_classes, 0);

    for (int i = 0; i < num_instances; ++i) {
        int trueClass = static_cast<int>(
            dataset->get_instance(i)->get(dataset->num_attributes() - 1)->operator float());
        int predictedClass = predictions[i];
        confusionMatrix[trueClass * num_classes + predictedClass]++;
    }

    return confusionMatrix;
}

float computeAccuracy(const vector<int>& confusionMatrix, ArffData* dataset) {
    int num_classes = dataset->num_classes();
    int successfulPredictions = 0;

    for (int i = 0; i < num_classes; ++i) {
        successfulPredictions += confusionMatrix[i * num_classes + i];
    }

    return 100.0f * successfulPredictions / dataset->num_instances();
}

int main(int argc, char* argv[]) {
    if (argc != 4) {
        cerr << "Usage: ./program datasets/train.arff datasets/test.arff k" << endl;
        return 1;
    }
    int k = stoi(argv[3]);

    ArffParser parserTrain(argv[1]);
    ArffParser parserTest(argv[2]);
    ArffData* train = parserTrain.parse();
    ArffData* test = parserTest.parse();

    // Measure execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start);

    vector<int> predictions = KNN(train, test, k);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    // Compute metrics
    vector<int> confusionMatrix = computeConfusionMatrix(predictions, test);
    float accuracy = computeAccuracy(confusionMatrix, test);


    cout << "The " << k << "-NN classifier for " << test->num_instances()
         << " test instances and " << train->num_instances()
         << " train instances required " << milliseconds
         << " ms CPU time for single-thread. Accuracy was "
         << accuracy << "%" << endl;

    return 0;
}